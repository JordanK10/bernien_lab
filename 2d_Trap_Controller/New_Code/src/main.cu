#include "hip/hip_runtime.h"
/** Created by J.Kemp with the atom Array team, Bernien group (2018). Based on
  protocol developed by H.Levin with the Lukin Group (2016) --
**/

#include <iostream>
#include "Console.h"

using namespace std;

// static bool const CONNECT_TO_CAMERA_SERVER = false;
static bool connectToAWG = false;

int main(){

    int sw_buf_size = pow(4,6);
    int wt_freq = 100;
    int cycles = 400;
    int clock_rate = MEGA(sw_buf_size*wt_freq/cycles); //Use mathematica to find parameter combination that is integer
    // double bandwidth = 15;
    int gain = 32761;

      TrapControllerHandler trapControllerHandler(clock_rate, gain, KILO(wt_freq));
      AWGController awgController(clock_rate,SEQUENCE,KILO(sw_buf_size));

      // if (awgsController.isConnected())
        run2DConsole(trapControllerHandler, awgController);
      return 0;
}
