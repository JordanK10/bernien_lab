
#include "TrapControllerHandler.h"
#include <fstream>

/* Generate width+length trap controllers. The length primary TCs are centered
on the width axis, and create a trap per x-increment starting at the
lowest frequency (x-y axis). Same for width, but along y-x axis */
TrapControllerHandler::TrapControllerHandler( double sampleRate, double gain, int wt_freq){

statHandler.y = new TrapController(sampleRate, gain, true,wt_freq);
statHandler.x = new TrapController(sampleRate, gain, true,wt_freq);

  tchLen = 0;
  tchWid = 0;
}

vector<vector<double>> TrapControllerHandler::trapFrequencies() {

  vector<vector<double>> frequenciesList;

  frequenciesList.push_back(statHandler.x->trapFrequencies());
  frequenciesList.push_back(statHandler.y->trapFrequencies());

	return frequenciesList;
}

void TrapControllerHandler::resetForRearrangement() {
  statHandler.x->resetForRearrangement();
  statHandler.x->resetForRearrangement();
}

bool TrapControllerHandler::sanitizeTraps(double new_gain,
	bool shouldPrintTotalPower){

  if (!statHandler.x->sanitizeTraps(new_gain, shouldPrintTotalPower)){
    statHandler.x->traps = statHandler.x->previousTraps;
    return false;
  }

  if (!statHandler.y->sanitizeTraps(new_gain, shouldPrintTotalPower)){
    statHandler.y->traps = statHandler.y->previousTraps;
    return false;
  }
  return true;
}

void TrapControllerHandler::saveTraps(){

  statHandler.x->previousTraps = statHandler.x->traps;
  statHandler.y->previousTraps = statHandler.y->traps;


}

void TrapControllerHandler::printAvailableDefaultTrapConfigurations() {
	DIR *dir;
	struct dirent *epdf;


	dir = opendir("C:\\Users\\bernien_lab\\Documents\\bernien_lab\\2d_Trap_Controller\\New_Code\\bin\\DefaultTrapConfigurations");

	vector<string> filenames;

	if (dir != NULL) {
		cout << "Available default trap configurations:" << endl;
		while ((epdf = readdir(dir))) {
			if (epdf->d_name[0] != '.') {
				filenames.push_back(epdf->d_name);
			}
		}

		sort(filenames.begin(), filenames.end());
		for (int i = 0; i < filenames.size(); i++) {
			cout << " " << filenames[i] << endl;
		}
	} else {
		cout << "Unable to open file." << endl;
	}
}

//void TrapControllerHanlder::setModes(vector<Waveform> modes, bool x){
//
//    if(x)
//      xmodes = modes;
//    if(!x)
//      ymodes = modes;
//}

// Loads a trap configuration from a file
bool TrapControllerHandler::loadDefaultTrapConfiguration(std::string filename){

  //Opens filestream from a given file. First two inputs are L/wW
  ifstream config_file("C:\\Users\\Bernien_Lab\\Documents\\bernien_lab\\2d_Trap_Controller\\New_Code\\bin\\DefaultTrapConfigurations\\" + filename);

  if (!config_file.is_open()) {
	  cout << "Unable to open file: " << filename << endl;
	  printAvailableDefaultTrapConfigurations();
	  return false;
  }

  saveTraps();
  statHandler.x->traps.clear();
  statHandler.y->traps.clear();

  // for(int )

  config_file >> tchLen;  config_file >> tchWid;
  int numTokensParsed=0; int numLinesParsed = 0; int numGroupsParsed = 0;

  //Generates list of token sets by group size, then sends them to corresponding trap
  vector<vector<string>> tokenList(tchLen);
  string temp;
  bool err = true;
  //Inputs trap information untill EOF
  while (config_file >> temp){
    tokenList[numLinesParsed].push_back(temp);
    numTokensParsed ++;
    if (numTokensParsed == 3) {
      numTokensParsed = 0;
      numLinesParsed ++;
      if (numLinesParsed ==tchLen) {
        if(numGroupsParsed == 0)
          err = statHandler.x->loadDefaultTrapConfiguration(tokenList,tchLen);
         if(numGroupsParsed == 1)
            err &= statHandler.y->loadDefaultTrapConfiguration(tokenList,tchWid);
        numLinesParsed = 0;
        numGroupsParsed ++;
      }
    }
    if(!err){
      cout << "bad";
      return false;

  }}

  lastLoadedConfiguration = filename;
  config_file.close();
  xmodes = statHandler.x->generateModes();
  ymodes = statHandler.y->generateModes();


  size_t size = xmodes[0].size()*sizeof(short);
  short* tempMode = NULL;
  short* tempMode2 = NULL;
  short* tempMode3 = NULL;
  for(int i = 0;i<xmodes.size();i++){//load the xmodes into xmodesCuda, which is
                                    //located on the GPU. If using 2 devices, Also
                                    //load xmodesCuda2 onto the second device
      tempMode = NULL;
      tempMode2 = NULL;
      tempMode3 = NULL;
      tempMode2 = static_cast<short*>(malloc(size));
      for(int j = 0;j<xmodes[0].size();j++){
        tempMode2[j] = xmodes[i][j];
      }
      if(numDevices == 1){
        hipSetDevice(defaultDevice);
        err =  hipMalloc((void **)&tempMode, size); if(err != hipSuccess){cout << "Memory Allocation Error (modes)"<<endl;}
        err = hipMemcpy(tempMode,tempMode2,size,hipMemcpyHostToDevice); if(err != hipSuccess){cout << "Memory Transfer Error (modes)" << endl;}
        xmodesCuda.push_back(tempMode);
      }
      if(numDevices == 2){
        hipSetDevice(0);
        err = hipMalloc((void **)&tempMode, size); if(err != hipSuccess){cout << "Memory Allocation Error (modes)"<<endl;}
        err = hipMemcpy(tempMode,tempMode2,size,hipMemcpyHostToDevice); if(err != hipSuccess){cout << "Memory Transfer Error (modes)" << endl;}
        xmodesCuda.push_back(tempMode);
        hipSetDevice(1);
        err = hipMalloc((void **)&tempMode3, size); if(err != hipSuccess){cout << "Memory Allocation Error (modes)"<<endl;}
        err = hipMemcpy(tempMode3,tempMode2,size,hipMemcpyHostToDevice);if(err != hipSuccess){cout << "Memory Transfer Error (modes)" << endl;}else{free(tempMode2);}
        xmodesCuda2.push_back(tempMode3);
      }
  }
  // for(int i = 0;i<ymodes.size();i++){
  //     tempMode = NULL;
  //     tempMode2 = NULL;
  //     tempMode2 = static_cast<short*>(malloc(size));
  //     for(int j = 0;j<ymodes[0].size();j++){
  //       tempMode2[j] = ymodes[i][j];
  //     }
  //     err =  hipMalloc((void **)&tempMode, size);
  //     if(err != hipSuccess){cout << "Memory Allocation Error"<<endl;}
  //     err = hipMemcpy(tempMode,tempMode2,size,hipMemcpyHostToDevice);
  //     if(err != hipSuccess){cout << "Memory Transfer Error" << endl;}else{free(tempMode2);}
  //     ymodesCuda.push_back(tempMode);
  // }
  mode_len = ymodes[0].size();

  return true;

}

void TrapControllerHandler::printTraps(){

  cout << "x Axis Traps: \n";
  statHandler.x->printTraps();
  cout << "\ny Axis Traps: \n";
  statHandler.y->printTraps();

}

/* Assumes filenames are of the form N(A).txt, where N is the number of traps
and A is the spacing in MHz.*/
// int numTrapsForConfigurationName(string config_name) {
// 	int index_of_parens = config_name.find_first_of('(');
//
// 	if (index_of_parens < 0) {
// 		return -1;
// 	}
//
// 	string num_traps_substring = config_name.substr(0, index_of_parens);
//
// 	return stoi(num_traps_substring);
// }

void TrapControllerHandler::initializeFromBinaryFile(string binaryFilename) {
	string path(dir);
	path.append(binaryFilename);

	ifstream binaryFile(path, ios::in | ios::binary);
	unsigned int numElements;
	vector<complex<float>> dataVector;
	vector<complex<float>> tempVector;
	binaryFile.read(reinterpret_cast<char*>(&numElements), sizeof(unsigned int));

	dataVector.resize(numElements);
	binaryFile.read(reinterpret_cast<char*>(&dataVector[0]), numElements * sizeof(dataVector[0]));

  for(int i=0; i<dataVector.size(); i++){
    break;
  }
}

bool fileExists(string filename) {
	string path(dir);
	path.append(filename);

	ifstream file(path, ios::in | ios::binary);
	return file.good();
}

bool TrapControllerHandler::initializeFromStaticWaveform(string trap_configuration_file) {
	int length = trap_configuration_file.length();
	string waveform_filename = trap_configuration_file.substr(0, length - 4) + "_static"; // Remove .txt, replace with _static.

	string waveform_path = static_waveforms + waveform_filename;

	if (fileExists(waveform_path)) {
		initializeFromBinaryFile(waveform_path);
		return true;
	} else {
		return false;
	}
}

// Iterates through all trapcontrollers, generating a list of waveforms to
//  to return
std::vector<Waveform> TrapControllerHandler::generateStaticWaveform(){

  std::vector<Waveform> wfList;

  //Generate x,y component of static wave
  wfList.push_back(statHandler.x->generateWaveform());
  wfList.push_back(statHandler.x->generateWaveform());

  return wfList;
}

void TrapController::printAvailableDefaultTrapConfigurations() {
	DIR *dir;
	struct dirent *epdf;


	dir = opendir("./DefaultTrapConfigurations");

	vector<string> filenames;

	if (dir != NULL) {
		cout << "Available default trap configurations:" << endl;
		while ((epdf = readdir(dir))) {
			if (epdf->d_name[0] != '.') {
				filenames.push_back(epdf->d_name);
			}
		}

		sort(filenames.begin(), filenames.end());
		for (int i = 0; i < filenames.size(); i++) {
			cout << " " << filenames[i] << endl;
		}
	} else {
		cout << "Unable to open file." << endl;
	}
}
// bool TrapController::mostRecentlyLoadedCorrectWaveforms(double duration, vector<RearrangementMove> moves) {
//
//   for(int i=0;i<moves.size();i++){
//     if(mostRecentlyLoadedCorrectWaveforms(1000,moves[i].startingConfig,moves[i].endingConfig))
//       return false;
//   }
//   return true;
// }
string dimensionFormat(string str, string ins){
	int lind = str.find('x');
	str.erase(lind, str.find('t')-lind);
	str.insert(lind, ins);
  return str;
}

bool TrapControllerHandler::loadPrecomputedWaveforms(double moveDuration, string startConfig, string endConfig) {

  // if(!statHandler.y->loadPrecomputedWaveforms(moveDuration,dimensionFormat(startConfig,"Y"), dimensionFormat(startConfig,"Y")))
  //   return false;

  if(!statHandler.x->loadPrecomputedWaveforms(moveDuration,dimensionFormat(startConfig,"X"), dimensionFormat(startConfig,"X")))
    return false;

  return true;
}

void TrapControllerHandler::cleanCudaModes(){//free modes from the GPU memory
  for(int i = 0;i<tchLen;i++){
    if(numDevices == 1){
      hipSetDevice(defaultDevice);
      hipFree(xmodesCuda[i]);
    }
    if(numDevices == 2){
      hipSetDevice(0);
      hipFree(xmodesCuda[i]);
      hipSetDevice(1);
      hipFree(xmodesCuda2[i]);
    }
  }
}

int TrapControllerHandler::rearrangeWaveforms(vector <RearrangementMove> moves, rearrange_mode mode, short* pvBuffer, short* cudaBuffer,short* cudaBuffer1) {
    const size_t movingWaveformSize = statHandler.x->getWFSize();
    int num_moves = moves.size();
    size_t size;
    size_t size1;
    hipError_t err = hipSuccess;
    auto start = chrono::high_resolution_clock::now();

    if(numDevices == 2){ //divide the moves in half if using 2 GPUs
      int n = num_moves/2 + num_moves%2;
      size1 = movingWaveformSize*n*sizeof(short)*2;
      size = movingWaveformSize*(num_moves - n)*sizeof(short)*2;
    }else{
      size = movingWaveformSize*num_moves*sizeof(short)*2;
    }

    if(numDevices == 2){
      int k = 0;
      for(int i=0; i<num_moves;i++){    //all data is now in statHandler.x
                                        //since statHandler.y is a duplicate
        if(i>=num_moves/2){
          statHandler.x->combinePrecomputedWaveform(moves[i].endingConfig, (xmodesCuda2[moves[i].dim]),k,cudaBuffer1,moves[i].row,mode_len, movingWaveformSize, num_moves,1);
          k++;
        }else{
          statHandler.x->combinePrecomputedWaveform(moves[i].endingConfig, (xmodesCuda[moves[i].dim]),i,cudaBuffer,moves[i].row,mode_len, movingWaveformSize, num_moves,0);
        }
      }
      err = hipSetDevice(0); if(err != hipSuccess){cout << "Device Set Error" << endl;}
      err = hipMemcpyAsync(pvBuffer,cudaBuffer,size,hipMemcpyDeviceToHost); if(err != hipSuccess){cout << "Mem transfer error: " << hipGetErrorString(err) << endl;}
      err = hipSetDevice(1); if(err != hipSuccess){cout << "Device Set Error: " << hipGetErrorString(err) << endl;}
      err = hipMemcpyAsync(&pvBuffer[size/sizeof(short)],cudaBuffer1,size1,hipMemcpyDeviceToHost); if(err != hipSuccess){cout << "Mem transfer error: " << hipGetErrorString(err) << endl;}
    }

    if(numDevices == 1){
      for(int i=0; i<num_moves;i++){    //all data is now in statHandler.x
                                        //since statHandler.y is a duplicate
        statHandler.x->combinePrecomputedWaveform(moves[i].endingConfig, (xmodesCuda[moves[i].dim]),i,cudaBuffer,moves[i].row,mode_len, movingWaveformSize, num_moves,defaultDevice);
      }
      err = hipSetDevice(defaultDevice); if(err != hipSuccess){cout << "Device Set Error" << endl;}
      err = hipMemcpyAsync(pvBuffer,cudaBuffer,size,hipMemcpyDeviceToHost); if(err != hipSuccess){cout << "Mem transfer error: " << hipGetErrorString(err) << endl;}
    }
    hipDeviceSynchronize();

    double dur = chrono::duration_cast<chrono::milliseconds>(chrono::high_resolution_clock::now() - start).count();
    cout << "Data Calculation Rate: " << movingWaveformSize*num_moves*2*sizeof(short)/dur/1e6 << " GB/s" << endl;

    // Transfer pvBuffer To Text File:

    // cout << "\n\nTransferring data to text file\n" << endl;
    // ofstream myFile;
    // ofstream myFile2;
    // myFile.open("c:/users/bernien_lab/desktop/sample_move_ch1.txt");
    // myFile2.open("c:/users/bernien_lab/desktop/sample_move_ch2.txt");
    // for(int k = 0;k<movingWaveformSize*num_moves;k++){
    //   myFile << pvBuffer[2*k] << endl;
    //   myFile2 << pvBuffer[2*k+1] << endl;
    // }
    // myFile.close();
    // myFile2.close();
    // cout << "\nTransfer Complete\n\n" << endl;

    // Print Samples from the pvBuffer:

    // for(int k = 0;k<movingWaveformSize*num_moves*2;k+=2000000){
    //   cout << k << " " << pvBuffer[k] << endl;
    // }

	return statHandler.x->getWFSize();
}
