#include "hip/hip_runtime.h"
/** Created by J.Kemp with the atom Array team, Bernien group (2018). Based on
  protocol developed by H.Levin with the Lukin Group (2016)
**/

#include "TrapController.h"
#include <iostream>

using namespace std;

TrapController::TrapController(double sampleRate,	double g, bool axis, int wt_freq) {
	srand(time(NULL));

  //The frequency of the wavetable
	long int waveTableFreq = wt_freq;

	waveTable = new WaveTable((long int)sampleRate, waveTableFreq);

	majorAxisx = axis;

	gain = g;

}

void TrapController::setCenter(double xaxis, double freq){
    if(xaxis)
        xAxisCenterFreq = freq;

    else
        yAxisCenterFreq = freq;

}


void TrapController::addTrap(double frequency, double amplitude, double phase) {
	traps.push_back(Trap(waveTable, frequency, amplitude, phase));
}



bool TrapController::loadDefaultTrapConfiguration(std::vector<std::vector<string>> tokenList, int groupSize) {

	vector<Trap> previousTraps = traps;

	traps.clear();
	string line;

	double freqx;
	// double freqy;
	double amplitude;
	double phase;

	// int numTokensParsed = 0;
	for(int i=0;i<groupSize; i++){
		vector<string> tokens  = tokenList[i];

		try {
			freqx = stod(tokens[0]) * 1.0E6;
			amplitude = stod(tokens[1])*gain;

			phase = stod(tokens[2]);

			addTrap(freqx, amplitude, phase);

		} catch (const invalid_argument&) {
			cout << "Invalid trap data";
		}
	}

	if (!sanitizeTraps(gain, false)) {
		cout << "Unable to load trap configuration: not sanitized." << endl;
		traps = previousTraps;
		return false;
	}

	return true;

}

Waveform TrapController::generateWaveform(double duration) {
	size_t num_samples = (size_t)(duration * waveTable->sampleRate);

	std::vector<short> waveform;
	waveform.resize(num_samples);

	for (size_t sample_index = 0; sample_index < num_samples; sample_index++) {
		float sample = 0;
		for (int trap_index = 0; trap_index < traps.size(); trap_index++)
			sample += real(traps[trap_index].nextSample());

		waveform[sample_index] = short(sample);
	}

	staticStartingWaveform = Waveform(waveform);
	return staticStartingWaveform;
}

vector<vector<short>> TrapController::generateModes() {
	size_t num_samples;

	vector<short> tempWaveform;
	vector<vector<short>> waveforms;

	for (int trap_index = 0; trap_index < traps.size(); trap_index++) {
	 	tempWaveform.clear();
		num_samples = (size_t)(1000*waveTable->tableLength*waveTable->tableFrequency/traps[trap_index].frequency);
		for (size_t sample_index = 0; sample_index < num_samples; sample_index++)
					tempWaveform.push_back((short)(real(traps[trap_index].nextSample())));
		waveforms.push_back(tempWaveform);
	}

	return waveforms;
}

//Checks to see if the traps are acceptable
bool TrapController::sanitizeTraps(double new_gain,
	bool shouldPrintTotalPower) {
	double totalPower = 0.0;
	double totalAmplitude = 0.0;

	for (int i = 0; i < traps.size(); i++) {
		double phase = traps[i].getPhase();
		if (phase < 0.0 || phase >= 10.0) {
			cout << "Trap #" << i << ": phase " << phase << " out of bounds [0, 1)" << endl;
			return false;
		}

		double freq = traps[i].frequency / 1.0E6;
		if (freq < 75 || freq > 105) {
			cout << "Trap #" << i << ": frequency " << freq << " out of bounds [75, 105]" << endl;
			return false;
		}

		double amp = traps[i].amplitude;
		if (amp < 0 || amp > 32767) {
			cout << "Trap #" << i << ": amplitude" << amp << " out of bounds [0, 32767]" << endl;
			return false;
		}

		double powerInMode = amp * amp; // Conversion from amplitude -> RF power
		totalPower += powerInMode;
		totalAmplitude += amp;
	}

	if (new_gain == -1) {
		new_gain = gain;
	}
	double gainFactor = pow(10.0, (new_gain - 1.0)/ 10.0);

	if (shouldPrintTotalPower) {
		cout << "Total power: " << totalPower * gainFactor << " mW" << endl;
	}

	// if (totalPower * gainFactor > 1600) { // Upper limit on power we can deliver to the AOD.
	// 	cout << "Total power " << totalPower * gainFactor << " out of bounds: must be < 1400 mW." << endl;
	// 	return false;
	// }

	// if (totalAmplitude > 0.99) { // Upper limit on total amplitude.
	// 	cout << "Total amplitude " << totalAmplitude << " out of bounds: must be <= 0.99" << endl;
	// 	return false;
	// }

	return true;
}

void TrapController::printTraps() {

	cout << traps.size() << "\n";
	for (int i = 0; i < traps.size(); i++) {
		cout << i << " " << traps[i].frequency << " " << traps[i].amplitude  << " " << traps[i].phase << endl;
	}
}

vector<double> TrapController::trapFrequencies() {
	vector<double> frequencies;
	for (int i = 0; i < traps.size(); i++) {
		frequencies.push_back(traps[i].frequency);
	}
	return frequencies;
}

void TrapController::resetForRearrangement() {
	// const size_t movingWaveformSize = rearrangeDataSize;
	// memset((void*)(rearrangeWaveform.dataShort[0]), 0, movingWaveformSize * sizeof(short));
	return;
	//STUB FIGURE OUT THIS
}


bool TrapController::mostRecentlyLoadedCorrectWaveforms(double duration, string starting_configuration, string ending_configuration) {
	if (lastLoadedWaveformProperties.starting_configuration.compare(starting_configuration) != 0) {
		return false;
	}

	if (lastLoadedWaveformProperties.ending_configuration.compare(ending_configuration) != 0) {
		return false;
	}

	if (lastLoadedWaveformProperties.duration != duration) {
		return false;
	}

	return true;
}

//set row to 0 if this is a row move, or row to 1 if this is a column move
__global__ void addWaveformsCuda(short* wave1, short* wave2, int row, int col,bool addMode, size_t movingWaveformSize, int startIndex, int endIndex,short* mode,int mode_len){
	//row and col are int values of either 0 or 1, and define how the mulitplexing is carried out.
	//If row is 1, col is 0 and vice waveform_on_samples
	//addMode is a bool that is true only on the first pass through a set of data
	//so that the modes are not repeatedly reassigned
	// this is the only kernel we have that executes on the GPU
	int i = blockDim.x * blockIdx.x + threadIdx.x + startIndex;
	wave1[i*2 + row] += wave2[i%movingWaveformSize]/3;
	if(addMode){
		wave1[i*2 + col] = mode[i%mode_len];
	}
}

void TrapController::combineRearrangeWaveformCuda(vector<int> *destinations, const size_t movingWaveformSize, short* mode, short* cudaBuffer, bool row, int mode_len, int num_moves, int move_index,int device) {

	int dest_index; int trap_index; short* dataArr;
	int threadsPerBlock = 128; //this should be either 128,256,512,or 1024
	int numBlocks = movingWaveformSize/threadsPerBlock; //the total nmumber of blocks*the number of threads
																											//per block has to be high enough to cycle though the whole
																											//move on the buffer
	int startIndex = movingWaveformSize*move_index;
	int endIndex = movingWaveformSize*(move_index + 1);
	hipError_t err;
	vector<vector<short*>> data;
	if(numDevices == 2){
		if(device == 1){
			err = hipSetDevice(1); if(err != hipSuccess){cout << "Device Set Error" << endl;}
			data = loadedCudaWaveforms2; //if there are 2 devices, the first half of the moves
																															//are run on the first device, the second half on the second
																															//loadedCudaWaveforms are on device 0, loadedCudaWaveforms2
																															//are on device 1
		}else{
			err = hipSetDevice(0); if(err != hipSuccess){cout << "Device Set Error" << endl;}
			data = loadedCudaWaveforms;
		}
	}
	if(numDevices == 1){
			err = hipSetDevice(defaultDevice); if(err != hipSuccess){cout << "Device Set Error" << endl;}
			data = loadedCudaWaveforms;
	}
	bool addMode = true;
	for (trap_index = 0; trap_index < destinations->size(); trap_index++) {
		dest_index = (*destinations)[trap_index];
		if (dest_index == -1) {
			continue;
		}
		dataArr = data[trap_index][dest_index];
		if(row){
			//invoke the Kernel
			addWaveformsCuda<<<numBlocks,threadsPerBlock>>>(cudaBuffer,dataArr,0,1,addMode,movingWaveformSize,startIndex,endIndex,mode,mode_len);
		}else{
				//invoke the Kernel
			addWaveformsCuda<<<numBlocks,threadsPerBlock>>>(cudaBuffer,dataArr,1,0,addMode,movingWaveformSize,startIndex,endIndex,mode,mode_len);
			}
		addMode = false;
		}
}


void TrapController::combineRearrangeWaveform(int worker, vector<int> *destinations, const size_t movingWaveformSize, std::vector<short> *mode, int move_block, short* pvBuffer, bool row, int mode_len, int bufferSize) {
	int chunkSize = movingWaveformSize / numWorkers;
	int startIndex = (chunkSize * worker)+(move_block);
	int endIndex = (chunkSize * (worker + 1)) + (move_block) ;

	int trap_index; int dest_index; int sample_index; short* dataArr;
	// auto start = chrono::high_resolution_clock::now(); //start

	for (trap_index = 0; trap_index < destinations->size(); trap_index++) {
		dest_index = (*destinations)[trap_index];
		dataArr = loadedTrapWaveforms[trap_index][dest_index].dataShort;

		if (dest_index == -1) {
			continue;
		}
		if(row){
			for (sample_index = startIndex; sample_index < endIndex; sample_index++)
				pvBuffer[sample_index*2] += dataArr[sample_index%movingWaveformSize]/3;
		}
		else{
			for (sample_index = startIndex; sample_index < endIndex; sample_index++)
				pvBuffer[sample_index*2+1] += dataArr[sample_index%movingWaveformSize]/3;
		}
	}
	if(row){
		for(sample_index = startIndex; sample_index < endIndex; sample_index++){
			pvBuffer[sample_index*2+1] += (*mode)[sample_index%mode_len];
		}
	}else{
		for(sample_index = startIndex; sample_index < endIndex; sample_index++){
			pvBuffer[sample_index*2] += (*mode)[sample_index%mode_len];
		}
	}

	// cout << chrono::duration_cast<chrono::milliseconds>(chrono::high_resolution_clock::now() - start).count() << "ms to combine move" << endl;

}

/* Moving traps: This will be the sum of the "loaded trap" waveforms for each
moving trap, designated by a start position and end position.
*/
void TrapController::combinePrecomputedWaveform(vector<int> &destinations, short* mode, int move_ind, short* pvBuffer, bool row, int mode_len,const size_t movingWaveformSize, int num_moves,int device){
	combineRearrangeWaveformCuda(&destinations, movingWaveformSize, mode, pvBuffer, row, mode_len, num_moves, move_ind, device);
	return;
	// thread *workers[numWorkers];
	// int mode_len = mode.size();
	// // Moving traps:
	// int block_ind = move_ind*movingWaveformSize;
	//
	// // Add each moving waveform separately.
	// for (int worker = 0; worker < numWorkers; worker++) {
	// 	workers[worker] = new thread(&TrapController::combineRearrangeWaveform, this, worker, &destinations, movingWaveformSize, &mode, block_ind, pvBuffer, row, mode_len, bufferSize);
	// }
	//
	// // Wait for all workers to finish combining waveforms.
	// for (int worker = 0; worker < numWorkers; worker++) {
	// 	workers[worker]->join();
	// }
}


int numTrapsForConfigurationName(string config_name) {
	int index_of_parens = config_name.find_first_of('(');
	if (index_of_parens < 0) {
		return -1;
	}

	string num_traps_substring = config_name.substr(0, index_of_parens);

	return stoi(num_traps_substring);
}

bool TrapController::loadPrecomputedWaveforms(double moveDuration, string starting_configuration, string ending_configuration) {

	// Use configuration names (up to first open parens) to determine the number of traps before and after rearrangement.
	numStartingTraps = numTrapsForConfigurationName(starting_configuration);
	numEndingTraps = numTrapsForConfigurationName(ending_configuration);

	if (numStartingTraps <= 0 || numEndingTraps <= 0) {
		cout << "Error: starting with " << numStartingTraps << " and ending with " << numEndingTraps << "!" << endl;
		cout << "Aborting!!!" << endl;

		return false;
	}


	cout << "Loading precomputed waveforms from " << numStartingTraps << " traps to " << numEndingTraps << " traps." << endl;
	cout << "\t" << fixed << setprecision(1) << moveDuration << " ms move time" << endl << endl;

	chrono::high_resolution_clock::time_point start_timer = chrono::high_resolution_clock::now();
	short* tempWave = NULL;
	// Rearranging waveforms:
	vector<short*> tempCudaWaveforms;
	vector<short*> tempCudaWaveforms2;
	hipError_t err;
	//load the precompute moving waveforms onto the host, and then move them
	//onto the GPU. If using 2 devices, load them all onto both devices. This
	//is rather inefficient in memory usage, but allows the fastest comp time
	//if the memory is available. once the mem is not available, we will need
	// to be trickier with memory usage, and how/where it is put
	for (int start_index = 0; start_index < numStartingTraps; start_index++) {
		for (int dest_index = 0; dest_index < numEndingTraps; dest_index++) {
			rearrangeDataSize = loadedTrapWaveforms[start_index][dest_index].initializeFromMovingWaveform(moveDuration,starting_configuration, ending_configuration,start_index, dest_index);
			tempWave = NULL;
			size_t size = rearrangeDataSize*sizeof(short);
			cout <<size/1E6<< " Megabytes" << endl;
			if(numDevices == 1){
				cout << 1 << endl;
				hipSetDevice(defaultDevice);
				err =  hipMalloc((void **)&tempWave, size);
				if(err != hipSuccess){cout << "Memory Allocation Error"<<endl;}
				err = hipMemcpy(tempWave,loadedTrapWaveforms[start_index][dest_index].dataShort,size,hipMemcpyHostToDevice);
				if(err != hipSuccess){cout << "Memory Transfer Error" << endl;}
				tempCudaWaveforms.push_back(tempWave);
			}

			if(numDevices == 2){
				cout << 2<< endl;
				hipSetDevice(0);
				err =  hipMalloc((void **)&tempWave, size);
				if(err != hipSuccess){cout << "Memory Allocation Error"<<endl;
					cout << hipGetErrorString(hipGetLastError()) << endl;
				}

				err = hipMemcpy(tempWave,loadedTrapWaveforms[start_index][dest_index].dataShort,size,hipMemcpyHostToDevice);
				if(err != hipSuccess){cout << "Memory Transfer Error" << endl;}
				tempCudaWaveforms.push_back(tempWave);

				hipSetDevice(1);
				tempWave = NULL;
				err =  hipMalloc((void **)&tempWave, size);
				if(err != hipSuccess){cout << "Memory Allocation Error"<<endl;}
				err = hipMemcpy(tempWave,loadedTrapWaveforms[start_index][dest_index].dataShort,size,hipMemcpyHostToDevice);
				if(err != hipSuccess){cout << "Memory Transfer Error" << endl;}
				tempCudaWaveforms2.push_back(tempWave);
			}
			 //loadedTrapWaveformsShort[start_index][dest_index].initializeShortFromFloatWaveform(loadedTrapWaveforms[start_index][dest_index].dataVector);
		}
		loadedCudaWaveforms.push_back(tempCudaWaveforms);
		tempCudaWaveforms = {};
		if(numDevices == 2){
			loadedCudaWaveforms2.push_back(tempCudaWaveforms2);
			tempCudaWaveforms2 = {};
		}
	}

	// for(int k = 0;k<1000;k++){
	// 	cout << loadedTrapWaveforms[6][0].dataShort[k] << endl;
	// }


	chrono::high_resolution_clock::time_point end_timer = chrono::high_resolution_clock::now();
	double msElapsed = (chrono::duration_cast<chrono::milliseconds>(chrono::high_resolution_clock::now() - start_timer)).count();

	cout << "\rLoading precomputed waveforms... done! (Time elapsed: " << (msElapsed / 1000.0) << " s)                      " << endl;

	if (rearrangeDataSize == 0) {
		cout << "Unable to read waveforms in from disk!" << endl;
		cout << "Aborting!!!" << endl;
		return false;
	}
	cout << "Waveform size: " << rearrangeDataSize << " samples." << endl;
	return true;
}


int TrapController::getWFSize(){
	return  rearrangeDataSize;
}
