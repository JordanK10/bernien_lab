#include "hip/hip_runtime.h"
/** Created by J.Kemp with the atom Array team, Bernien group (2018). Based on
  protocol developed by H.Levin with the Lukin Group (2016)
**/

#include "TrapController.h"
#include <iostream>

using namespace std;

TrapController::TrapController(double sampleRate,	double g, bool axis, int wt_freq) {
	srand(time(NULL));

  //The frequency of the wavetable
	long int waveTableFreq = wt_freq;

	waveTable = new WaveTable((long int)sampleRate, waveTableFreq);

	majorAxisx = axis;

	gain = g;

}

void TrapController::setCenter(double xaxis, double freq){
    if(xaxis)
        xAxisCenterFreq = freq;

    else
        yAxisCenterFreq = freq;

}


void TrapController::addTrap(double frequency, double amplitude, double phase) {
	traps.push_back(Trap(waveTable, frequency, amplitude, phase));
}



bool TrapController::loadDefaultTrapConfiguration(std::vector<std::vector<string>> tokenList, int groupSize) {

	vector<Trap> previousTraps = traps;

	traps.clear();
	string line;

	double freqx;
	// double freqy;
	double amplitude;
	double phase;

	// int numTokensParsed = 0;
	for(int i=0;i<groupSize; i++){
		vector<string> tokens  = tokenList[i];

		try {
			freqx = stod(tokens[0]) * 1.0E6;
			amplitude = stod(tokens[1])*gain;

			phase = stod(tokens[2]);

			addTrap(freqx, amplitude, phase);

		} catch (const invalid_argument&) {
			cout << "Invalid trap data";
		}
	}

	if (!sanitizeTraps(gain, false)) {
		cout << "Unable to load trap configuration: not sanitized." << endl;
		traps = previousTraps;
		return false;
	}

	return true;

}

Waveform TrapController::generateWaveform(double duration) {
	size_t num_samples = (size_t)(duration * waveTable->sampleRate);

	std::vector<short> waveform;
	waveform.resize(num_samples);

	for (size_t sample_index = 0; sample_index < num_samples; sample_index++) {
		float sample = 0;
		for (int trap_index = 0; trap_index < traps.size(); trap_index++)
			sample += real(traps[trap_index].nextSample());

		waveform[sample_index] = short(sample);
	}

	staticStartingWaveform = Waveform(waveform);
	return staticStartingWaveform;
}

vector<vector<short>> TrapController::generateModes() {
	size_t num_samples;

	vector<short> tempWaveform;
	vector<vector<short>> waveforms;

	for (int trap_index = 0; trap_index < traps.size(); trap_index++) {
	 	tempWaveform.clear();
		num_samples = (size_t)(1000*waveTable->tableLength*waveTable->tableFrequency/traps[trap_index].frequency);
		for (size_t sample_index = 0; sample_index < num_samples; sample_index++)
					tempWaveform.push_back((short)(real(traps[trap_index].nextSample())));
		waveforms.push_back(tempWaveform);
	}

	return waveforms;
}

//Checks to see if the traps are acceptable
bool TrapController::sanitizeTraps(double new_gain,
	bool shouldPrintTotalPower) {
	double totalPower = 0.0;
	double totalAmplitude = 0.0;

	for (int i = 0; i < traps.size(); i++) {
		double phase = traps[i].getPhase();
		if (phase < 0.0 || phase >= 10.0) {
			cout << "Trap #" << i << ": phase " << phase << " out of bounds [0, 1)" << endl;
			return false;
		}

		double freq = traps[i].frequency / 1.0E6;
		if (freq < 75 || freq > 105) {
			cout << "Trap #" << i << ": frequency " << freq << " out of bounds [75, 105]" << endl;
			return false;
		}

		double amp = traps[i].amplitude;
		if (amp < 0 || amp > 32767) {
			cout << "Trap #" << i << ": amplitude" << amp << " out of bounds [0, 32767]" << endl;
			return false;
		}

		double powerInMode = amp * amp; // Conversion from amplitude -> RF power
		totalPower += powerInMode;
		totalAmplitude += amp;
	}

	if (new_gain == -1) {
		new_gain = gain;
	}
	double gainFactor = pow(10.0, (new_gain - 1.0)/ 10.0);

	if (shouldPrintTotalPower) {
		cout << "Total power: " << totalPower * gainFactor << " mW" << endl;
	}

	// if (totalPower * gainFactor > 1600) { // Upper limit on power we can deliver to the AOD.
	// 	cout << "Total power " << totalPower * gainFactor << " out of bounds: must be < 1400 mW." << endl;
	// 	return false;
	// }

	// if (totalAmplitude > 0.99) { // Upper limit on total amplitude.
	// 	cout << "Total amplitude " << totalAmplitude << " out of bounds: must be <= 0.99" << endl;
	// 	return false;
	// }

	return true;
}

void TrapController::printTraps() {

	cout << traps.size() << "\n";
	for (int i = 0; i < traps.size(); i++) {
		cout << i << " " << traps[i].frequency << " " << traps[i].amplitude  << " " << traps[i].phase << endl;
	}
}

vector<double> TrapController::trapFrequencies() {
	vector<double> frequencies;
	for (int i = 0; i < traps.size(); i++) {
		frequencies.push_back(traps[i].frequency);
	}
	return frequencies;
}

void TrapController::resetForRearrangement() {
	// const size_t movingWaveformSize = rearrangeDataSize;
	// memset((void*)(rearrangeWaveform.dataShort[0]), 0, movingWaveformSize * sizeof(short));
	return;
	//STUB FIGURE OUT THIS
}


bool TrapController::mostRecentlyLoadedCorrectWaveforms(double duration, string starting_configuration, string ending_configuration) {
	if (lastLoadedWaveformProperties.starting_configuration.compare(starting_configuration) != 0) {
		return false;
	}

	if (lastLoadedWaveformProperties.ending_configuration.compare(ending_configuration) != 0) {
		return false;
	}

	if (lastLoadedWaveformProperties.duration != duration) {
		return false;
	}

	return true;
}

//set row to 0 if this is a row move, or row to 1 if this is a column move
__global__ void addWaveformsCuda(short* wave1, short* wave2, int row, int col,bool addMode, size_t movingWaveformSize, int startIndex, int endIndex,short* mode,int mode_len){
	int i = blockDim.x * blockIdx.x + threadIdx.x + startIndex;
	if(i >= startIndex && i<endIndex){
		wave1[i*2 + row] += wave2[i%movingWaveformSize]/3;
		if(addMode){
			wave1[i*2 + col] = mode[i%mode_len];
		}
	}

	__syncthreads();
}

void TrapController::combineRearrangeWaveformCuda(vector<int> *destinations, const size_t movingWaveformSize, short* mode, short* cudaBuffer, bool row, int mode_len, int bufSize,int num_moves, int move_index) {

	int dest_index; int trap_index; short* dataArr;
	int threadsPerBlock = 128;
	int numBlocks = movingWaveformSize/threadsPerBlock;

	int startIndex = movingWaveformSize*move_index;
	int endIndex = movingWaveformSize*(move_index + 1);

	bool addMode = true;

	for (trap_index = 0; trap_index < destinations->size(); trap_index++) {
		dest_index = (*destinations)[trap_index];
		if (dest_index == -1) {
			continue;
		}
		dataArr = loadedCudaWaveforms[trap_index][dest_index];
		if(row){
			//invoke the Kernel
			addWaveformsCuda<<<numBlocks,threadsPerBlock>>>(cudaBuffer,dataArr,0,1,addMode,movingWaveformSize,startIndex,endIndex,mode,mode_len);
		}else{
				//invoke the Kernel
			addWaveformsCuda<<<numBlocks,threadsPerBlock>>>(cudaBuffer,dataArr,1,0,addMode, movingWaveformSize, startIndex, endIndex,mode,mode_len);
			}
			addMode = false;
		}
}


void TrapController::combineRearrangeWaveform(int worker, vector<int> *destinations, const size_t movingWaveformSize, std::vector<short> *mode, int move_block, short* pvBuffer, bool row, int mode_len, int bufferSize) {
	int chunkSize = movingWaveformSize / numWorkers;
	int startIndex = (chunkSize * worker)+(move_block);
	int endIndex = (chunkSize * (worker + 1)) + (move_block) ;

	int trap_index; int dest_index; int sample_index; short* dataArr;
	// auto start = chrono::high_resolution_clock::now(); //start

	for (trap_index = 0; trap_index < destinations->size(); trap_index++) {
		dest_index = (*destinations)[trap_index];
		dataArr = loadedTrapWaveforms[trap_index][dest_index].dataShort;

		if (dest_index == -1) {
			continue;
		}
		if(row){
			for (sample_index = startIndex; sample_index < endIndex; sample_index++)
				pvBuffer[sample_index*2] += dataArr[sample_index%movingWaveformSize]/3;
		}
		else{
			for (sample_index = startIndex; sample_index < endIndex; sample_index++)
				pvBuffer[sample_index*2+1] += dataArr[sample_index%movingWaveformSize]/3;
		}
	}
	if(row){
		for(sample_index = startIndex; sample_index < endIndex; sample_index++){
			pvBuffer[sample_index*2+1] += (*mode)[sample_index%mode_len];
		}
	}else{
		for(sample_index = startIndex; sample_index < endIndex; sample_index++){
			pvBuffer[sample_index*2] += (*mode)[sample_index%mode_len];
		}
	}

	// cout << chrono::duration_cast<chrono::milliseconds>(chrono::high_resolution_clock::now() - start).count() << "ms to combine move" << endl;

}

/* Moving traps: This will be the sum of the "loaded trap" waveforms for each
moving trap, designated by a start position and end position.
*/
void TrapController::combinePrecomputedWaveform(vector<int> &destinations, short* mode, int move_ind, short* pvBuffer, bool row, int mode_len,const size_t movingWaveformSize, int bufferSize, int num_moves){

	combineRearrangeWaveformCuda(&destinations, movingWaveformSize, mode, pvBuffer, row, mode_len, bufferSize, num_moves, move_ind);
	return;
	// thread *workers[numWorkers];
	// int mode_len = mode.size();
	// // Moving traps:
	// int block_ind = move_ind*movingWaveformSize;
	//
	// // Add each moving waveform separately.
	// for (int worker = 0; worker < numWorkers; worker++) {
	// 	workers[worker] = new thread(&TrapController::combineRearrangeWaveform, this, worker, &destinations, movingWaveformSize, &mode, block_ind, pvBuffer, row, mode_len, bufferSize);
	// }
	//
	// // Wait for all workers to finish combining waveforms.
	// for (int worker = 0; worker < numWorkers; worker++) {
	// 	workers[worker]->join();
	// }
}


int numTrapsForConfigurationName(string config_name) {
	int index_of_parens = config_name.find_first_of('(');
	if (index_of_parens < 0) {
		return -1;
	}

	string num_traps_substring = config_name.substr(0, index_of_parens);

	return stoi(num_traps_substring);
}

bool TrapController::loadPrecomputedWaveforms(double moveDuration, string starting_configuration, string ending_configuration) {

	// Use configuration names (up to first open parens) to determine the number of traps before and after rearrangement.
	numStartingTraps = numTrapsForConfigurationName(starting_configuration);
	numEndingTraps = numTrapsForConfigurationName(ending_configuration);

	if (numStartingTraps <= 0 || numEndingTraps <= 0) {
		cout << "Error: starting with " << numStartingTraps << " and ending with " << numEndingTraps << "!" << endl;
		cout << "Aborting!!!" << endl;

		return false;
	}


	cout << "Loading precomputed waveforms from " << numStartingTraps << " traps to " << numEndingTraps << " traps." << endl;
	cout << "\t" << fixed << setprecision(1) << moveDuration << " ms move time" << endl << endl;

	chrono::high_resolution_clock::time_point start_timer = chrono::high_resolution_clock::now();
	short* tempWave = NULL;
	// Rearranging waveforms:
	vector<short*> tempCudaWaveforms;
	hipError_t err;
	for (int start_index = 0; start_index < numStartingTraps; start_index++) {
		for (int dest_index = 0; dest_index < numEndingTraps; dest_index++) {
			rearrangeDataSize = loadedTrapWaveforms[start_index][dest_index].initializeFromMovingWaveform(moveDuration,starting_configuration, ending_configuration,start_index, dest_index);
			tempWave = NULL;
			size_t size = rearrangeDataSize*sizeof(short);

			err =  hipMalloc((void **)&tempWave, size);
			if(err != hipSuccess){cout << "Memory Allocation Error"<<endl;}

			err = hipMemcpy(tempWave,loadedTrapWaveforms[start_index][dest_index].dataShort,size,hipMemcpyHostToDevice);
			if(err != hipSuccess){cout << "Memory Transfer Error" << endl;}

			tempCudaWaveforms.push_back(tempWave);
			 //loadedTrapWaveformsShort[start_index][dest_index].initializeShortFromFloatWaveform(loadedTrapWaveforms[start_index][dest_index].dataVector);
		}
		loadedCudaWaveforms.push_back(tempCudaWaveforms);
		tempCudaWaveforms = {};
	}

	// for(int k = 0;k<1000;k++){
	// 	cout << loadedTrapWaveforms[6][0].dataShort[k] << endl;
	// }


	chrono::high_resolution_clock::time_point end_timer = chrono::high_resolution_clock::now();
	double msElapsed = (chrono::duration_cast<chrono::milliseconds>(chrono::high_resolution_clock::now() - start_timer)).count();

	cout << "\rLoading precomputed waveforms... done! (Time elapsed: " << (msElapsed / 1000.0) << " s)                      " << endl;

	if (rearrangeDataSize == 0) {
		cout << "Unable to read waveforms in from disk!" << endl;
		cout << "Aborting!!!" << endl;
		return false;
	}
	cout << "Waveform size: " << rearrangeDataSize << " samples." << endl;
	return true;
}


int TrapController::getWFSize(){
	return  rearrangeDataSize;
}
