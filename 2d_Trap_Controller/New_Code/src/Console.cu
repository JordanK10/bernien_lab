#include "hip/hip_runtime.h"
#include "Console.h"

double m_dur =1;

//#include "control_interface.h"

struct RearrangementMove;

std::vector<string> parseCommand(string &cmd) {
	if (cmd[cmd.size() - 1] == '\n') {
		cmd.pop_back();
	}

	std::vector<string> tokens;

	stringstream ss(cmd);
	string item;

	while (getline(ss, item, ' ')) {
		if (item.size() > 0) {
			tokens.push_back(item);
		}
	}

	return tokens;
}

static chrono::high_resolution_clock::time_point start_timer;
void startTimer() {
	start_timer = chrono::high_resolution_clock::now();
}

double timeElapsed() {
	return chrono::duration_cast<chrono::milliseconds>(chrono::high_resolution_clock::now() - start_timer).count();
}

std::vector<string> takeInput() {
	cout << "#: ";
	string input;
	getline(cin,input);

	if (input[input.size() - 1] == '\n') {
		input.pop_back();
	}

	return parseCommand(input);
}



int promptForInteger(string attr) {
	int val;
	cout << "Enter " << attr << ": ";
	while (!(cin >> val)) {
		cout << "Unable to parse value." << endl;
		cin.clear();
		cin.ignore(0xfffff, '\n');
		cout << "Enter " << attr << ": " << endl;
	}

	return val;
}

std::vector<int> promptForIntegerSequence(string attr) {

	while (true) {
		cout << "Enter " << attr << ": ";
		string input;
		getline(cin,input);
		std::vector<string> tokens = parseCommand(input);

		if (tokens.size() > 0) {
			bool allTokensValid = true;
			std::vector<int> intSequence;

			for (int i = 0; i < tokens.size(); i++) {
				try {
					int val = stoi(tokens[i]);
					intSequence.push_back(val);
				}
				catch (const std::invalid_argument&) {
					cout << "Unable to parse as integer: " << tokens[i] << endl;
					allTokensValid = false;
					break;
				}
			}

			if (allTokensValid) {
				return intSequence;
			}

		} else {
			cout << "Unable to parse cluster integer sequence. Example: 2 3 2" << endl;
		}
	}
}

void printTrapHelp() {
	cout << "Trap command usage list:" << endl;
	cout << "traps list" << endl;
	cout << "traps sort" << endl;
	cout << "traps add [freq] [amp]" << endl;
	cout << "traps change [index] [property: freq, amp, phase] [new value]" << endl;
	cout << "traps delete [index]" << endl;
	cout << "traps clean" << endl;
	cout << "traps load_default [default_name: 100(0.49).txt, 70(0.70).txt]" << endl;
	cout << "traps set_phases [phase arguments]" << endl;
}

void printCameraHelp() {
	cout << "Camera command usage list:" << endl;
	cout << "camera connect" << endl;
}

void printAWGHelp() {
	cout << "AWG command usage list:" << endl;
	cout << "awg connect" << endl;
	cout << "awg gain" << endl;
	cout << "awg gain [new value]" << endl;
	cout << "awg timeout" << endl;
	cout << "awg timeout [new value]" << endl;
	cout << "awg load_waveform [filename]" << endl;
}

void printRunHelp() {
	cout << "Run command usage list:" << endl;
	cout << "run rearrangement" << endl;
	cout << "run rearrangement [duration (ms)]" << endl;
	cout << "run rearrangement [fade duration] [move duration]" << endl;
}

void printAdwinHelp() {
	cout << "Adwin command usage list:" << endl;
	cout << "adwin send_trigger" << endl;
}

void pickRearrangementMode(enum rearrange_mode &mode, int &modeArgument) {
	cout << "Please select a rearrangement mode:" << endl;
	cout << "0: Arrang atoms in the Center." << endl;
	cout << "1: Stack from upper left corner." << endl;
	cout << "2: Stack from upper right corner." << endl;
	cout << "3: Stack from lower left corner." << endl;
	cout << "4: Stack from lower right corner." << endl;
	cout << "5: Stack from closest corner." << endl;
	cout << "6: Rectangular from the left." << endl;
	cout << "7: Rectangular from the right." << endl;
	cout << "8: Rectangular from the center." << endl;


	int enteredMode = -1;
	while (true) {
		enteredMode = promptForInteger("rearrangement mode");

		if (enteredMode >= 0 && enteredMode <= 8) {
			break;
		}
		cout << "Please pick a mode from 0 to 8." << endl;
	}
	mode = (enum rearrange_mode)enteredMode;

	// The mode argument is only used for options 2 and 3.
	// if (mode == REARRANGE_MODE_FIXED_ARRAY_WITHOUT_RESERVOIR || mode == REARRANGE_MODE_FIXED_ARRAY_WITH_RESERVOIR || mode == REARRANGE_MODE_FIXED_ARRAY_WITH_NEARBY_RESERVOIR) {
	// 	int enteredTargetNum = -1;
	//
	// 	while (true) {
	// 		enteredTargetNum = promptForInteger("target array size");
	//
	// 		if (enteredTargetNum >= 0) {
	// 			break;
	// 		}
	// 		cout << "Please pick a non-negative target size." << endl;
	// 	}
	//
	// 	modeArgument = enteredTargetNum;
	// }
}

void runRearrangementSequence(TrapControllerHandler &trapControllerHandler, AWGController &awgController,
							  double moveDuration, string starting_configuration, string ending_configuration) {

	if (trapControllerHandler.tchWid>0){
		cout << "Summoning Maxwell's demon..." << endl;
		cout << "Rearranging from " << starting_configuration << " configuration to " << ending_configuration << " in ";
		cout << fixed << setprecision(1) << moveDuration << " ms." << endl;

		rearrange_method method = BALANCE_COMPRESS;
		rearrange_mode mode;
		int modeArgument;
		pickRearrangementMode(mode, modeArgument);

		 if (!awgController.isConnected()) {
		 	cout << "Error: Not connected to AWG!" << endl;
		 	cout << "***Aborting sequence***" << endl;
			return;
		}

		// Start sending waveform to AWG.
		Waveform startingXWaveform = trapControllerHandler.staticXWaveform;
		Waveform startingYWaveform = trapControllerHandler.staticYWaveform;

		CameraServer cameraServer;
		cameraServer.startServer();

		while (true) {

			// Record durations.
			std::vector<double> durations;
			std::vector<bool> underflowRecords;

			// We push the static waveforms to the static traps
			awgController.pushStaticWaveforms(trapControllerHandler.generateStaticWaveform(),true);
			// #include <fstream>
			// ofstream myFile;
			// myFile.open("c:/users/bernien_lab/desktop/38_moves_comp_time_data.csv");
			// // Keeping track of number of rearrangements
			int numRearrangementsPerformed = 0;
			while (true) {
				// Find atoms in new picture on camera:
				// for(int i = 0;i<10000;i++){
				std::vector<std::vector<bool>> atomsPresent = cameraServer.receiveIdentifiedAtomList(trapControllerHandler.trapFrequencies().size(),trapControllerHandler.tchLen);
				if (atomsPresent.size() == 0) {
					cout << "Camera server returned an empty list. Aborting...\n";
					break;
				}

				numRearrangementsPerformed++;

				//Setting for rearrangement
				trapControllerHandler.resetForRearrangement();

				startTimer();
				vector<RearrangementMove> moves = rearrange(atomsPresent,method,mode);
				// RearrangementMove temp_move;
				// vector<RearrangementMove> moves;
				// temp_move.startingConfig = {1,1,1,1,1,1,1,1,1,1};
				// temp_move.endingConfig = {9,-1,-1,-1,-1,-1,-1,-1,-1,-1};
				// temp_move.row = false;
				// temp_move.dim = 0;
				// moves.push_back(temp_move);
				//
				// temp_move.startingConfig = {1,1,1,1,1,1,1,1,1,1};
				// temp_move.endingConfig = {-1,-1,-1,-1,-1,-1,-1,-1,-1,0};
				// temp_move.row = false;
				// temp_move.dim = 0;
				// moves.push_back(temp_move);

				int duration = timeElapsed();



				int move_len = trapControllerHandler.rearrangeWaveforms(moves,mode,awgController.getDynamicBuffer(),awgController.getCudaBuffer(),awgController.getCudaBuffer2());
				int duration2 = timeElapsed();
				 // myFile << moves.size() << "," << duration2-duration << endl;
				 // }
				awgController.pushRearrangeWaveforms(moves.size(),move_len);
				int duration3 = timeElapsed();
				cout << "Software rearrange time: " << duration << " ms" << endl;
				cout << "Waveform generation  time: " << duration2-duration << " ms" << endl; //this took 13 ms longer
				cout << "Pushing waveform time -> trigger: " << duration3-duration2-duration << " ms" << endl;
				durations.push_back(duration);
				break;
			}

			double avgDuration = 0;
			for (int i = 0; i < durations.size(); i++) {
				avgDuration += durations[i];
			}
			avgDuration /= (1.0 * durations.size());

			double durationVariance = 0;
			for (int i = 0; i < durations.size(); i++) {
				durationVariance += pow(avgDuration - durations[i], 2.0);
			}
			double stdDev = sqrt(durationVariance / durations.size());

			cout << "Duration from recv trigger -> send trigger: " << avgDuration << " +/- " << stdDev << endl;
			break;

		}
	}else{
		cout << "No traps present... Aborting\n";
	}

}

void processRunCommand(std::vector<string> &commandTokens, TrapControllerHandler &trapControllerHandler, AWGController &awgController) {
	if (commandTokens.size() == 1 || commandTokens[1].compare("help") == 0) {
		printRunHelp();
	} else if (commandTokens[1].compare("rearrangement") == 0) {

		double moveDuration = m_dur; //in milliseconds


		// Default name of trap configurations to start and end with.
		string starting_configuration = trapControllerHandler.lastLoadedConfiguration;
		string ending_configuration = trapControllerHandler.lastLoadedConfiguration;

		// If parameters passed to call, then replace the default values.
		for (int tokenIndex = 2; tokenIndex + 1 < commandTokens.size(); tokenIndex += 2) {
			string indicator = commandTokens[tokenIndex];
			string arg = commandTokens[tokenIndex + 1];

			if (indicator.compare("-d") == 0) {
				try {
					double dur = stod(arg);
					moveDuration = dur;
				}
				catch (const std::invalid_argument&) {
					cout << "Unable to parse duration!" << endl;
				}
			} else if (indicator.compare("-start") == 0) {
				starting_configuration = arg;
			} else if (indicator.compare("-end") == 0) {
				ending_configuration = arg;
			}
		}
		runRearrangementSequence(trapControllerHandler, awgController, moveDuration, starting_configuration, ending_configuration);
	} else {
		cout << "Run command unknown: " << commandTokens[1] << endl;
		printRunHelp();
	}
}

bool compareTrapFrequencies(Trap i, Trap j) {
	return i.frequency < j.frequency;
}

void processAWGInput(vector<string> &commandTokens, TrapControllerHandler &trapControllerHandler, AWGController &awgController) {
	if (commandTokens.size() == 1 || commandTokens[1].compare("help") == 0) {
		printAWGHelp();
	} else if (commandTokens[1].compare("connect") == 0) {
		if (awgController.isConnected()) {
			cout << "AWG already connected!" << endl;
		} else {
			awgController.pushStaticWaveforms(trapControllerHandler.generateStaticWaveform(),true);
		}
	} else if (commandTokens[1].compare("disconnect") == 0) {
		if (awgController.isConnected()) {
			awgController.disconnect();
		} else {
			cout << "Already disconnected!" << endl;
		}
	} else if (commandTokens[1].compare("gain") == 0) {
			cout << "There is currently no gain functionality\n";
	// 	if (commandTokens.size() == 2) {
	// 		cout << "AWG Gain: " << awgController.getGain() << endl;
	// 	} else if (commandTokens.size() >= 3) {
	// 		try {
	// 			double gain = stod(commandTokens[2]);
	// 			if (trapControllerHandler.sanitizeTraps(gain)) {
	//
	// 				bool success = awgController.changeGain(gain);
	//
	// 				if (success) {
	// 					cout << "Set AWG Gain to " << awgController.getGain() << endl;
	// 					trapControllerHandler.awg_gain = awgController.getGain();
	// 				} else {
	// 					cout << "Unable to change gain - not connected to AWG!" << endl;
	// 				}
	// 			} else {
	// 				cout << "Unable to change gain - too much power!" << endl;
	// 			}
	// 		} catch (const invalid_argument&) {
	// 			cout << "Unable to parse gain!" << endl;
	// 		}
	// 	}
	} else if (commandTokens[1].compare("load_waveform") == 0) {
		if (commandTokens.size() >= 3) {
			string filename = commandTokens[2];
			Waveform w(filename);

			// awgController.pushWaveform(w);
		} else {
			cout << "Usage: awg load_waveform [filename]" << endl;
			cout << "NOTE: Please make sure that the waveform is sanitized" << endl;
			cout << " and that the gain is good before loading a waveform." << endl;
		}
	} else if (commandTokens[1].compare("push_waveform") == 0) {
		awgController.pushStaticWaveforms(trapControllerHandler.generateStaticWaveform(),true);
	} else if (commandTokens[1].compare("push_trans_waveform") == 0) {
		awgController.pushStaticWaveforms(trapControllerHandler.generateStaticWaveform(),false);
	} else if (commandTokens[1].compare("trigger") == 0) {
		awgController.triggerSequence();
	}else if (commandTokens[1].compare("switch") == 0) {
		if(commandTokens[2].compare("fifo") == 0)
			awgController.changeMode(FIFO);
		else if(commandTokens[2].compare("single") == 0)
			awgController.changeMode(SINGLE);
	} else if (commandTokens[1].compare("run") == 0) {
		awgController.run(0,1);
		return;
	} else if (commandTokens[1].compare("stop") == 0){
		awgController.stop();
	}else {
		printAWGHelp();
	}
}

// Returns whether the waveform is different now.
bool processTrapsInput(std::vector<string> &commandTokens, TrapControllerHandler &trapControllerHandler, AWGController &awgController) {

	std::vector<std::vector<Trap>> previousTraps;

	previousTraps.push_back(trapControllerHandler.statHandler.x->traps);
	previousTraps.push_back(trapControllerHandler.statHandler.y->traps);

	bool waveformShouldChange = false;

	if (commandTokens.size() == 1 || commandTokens[1].compare("help") == 0) {
		printTrapHelp();
	} else if (commandTokens[1].compare("list") == 0) {
		trapControllerHandler.printTraps();
	} else if (commandTokens[1].compare("sort") == 0) {
			sort(trapControllerHandler.statHandler.x->traps.begin(), trapControllerHandler.statHandler.x->traps.end(), compareTrapFrequencies);
			trapControllerHandler.statHandler.x->printTraps();
			sort(trapControllerHandler.statHandler.y->traps.begin(), trapControllerHandler.statHandler.y->traps.end(), compareTrapFrequencies);
			trapControllerHandler.statHandler.y->printTraps();
	} else if (commandTokens[1].compare("add") == 0) {
		if (commandTokens.size() >= 4) {
			try {
				double freqx = stod(commandTokens[2]);
				int x = stod(commandTokens[3]);
				double freqy = stod(commandTokens[4]);
				int y = stod(commandTokens[5]);
				double ampl = stod(commandTokens[6]);
				trapControllerHandler.statHandler.x->addTrap(freqx * 1.0E6, ampl);
				waveformShouldChange = true;
			}
			catch (const invalid_argument&) {
				cout << "Unable to parse values as numbers!" << endl;
			}
		}
		else {
			cout << "Usage: add [freq in MHz] [amplitude]" << endl;
		}
	} else if (commandTokens[1].compare("delete") == 0) {
		if (commandTokens.size() >= 3) {
			try {
				int x = stod(commandTokens[3]);
				int y = stod(commandTokens[5]);
				if (x < 0 || y < 0 ||  y >= trapControllerHandler.statHandler.x->traps.size()) {
					cout << "Index out of range!" << endl;
				}
				else {
					trapControllerHandler.statHandler.x->traps.erase(	trapControllerHandler.statHandler.x->traps.begin() + x);
					waveformShouldChange = true;
				}
			}
			catch (const std::invalid_argument&) {
				cout << "Unable to parse index!" << endl;
			}
		}
		else {
			cout << "Usage: delete [trap index]" << endl;
		}
	} else if (commandTokens[1].compare("clean") == 0) {
		for(int i = 0;i<trapControllerHandler.tchLen;i++){
			for(int j = 0;j<trapControllerHandler.tchLen;j++){
				if(numDevices == 1){
					hipSetDevice(defaultDevice);
					hipFree(trapControllerHandler.statHandler.x->loadedCudaWaveforms[i][j]);
				}
				if(numDevices == 2){
					hipSetDevice(0);
					hipFree(trapControllerHandler.statHandler.x->loadedCudaWaveforms[i][j]);
					hipSetDevice(1);
					hipFree(trapControllerHandler.statHandler.x->loadedCudaWaveforms2[i][j]);
				}
			}
		}
		awgController.cleanCudaBuffer();
		trapControllerHandler.cleanCudaModes();

		for(int i=0; i<trapControllerHandler.tchLen; i++){
			trapControllerHandler.statHandler.x->traps.erase(	trapControllerHandler.statHandler.x->traps.begin());
		}
		// for(int i=0; i<trapControllerHandler.tchWid; i++){
		// 	trapControllerHandler.statHandler.y->traps.erase(	trapControllerHandler.statHandler.y->traps.begin());
		// }

	} else if (commandTokens[1].compare("change") == 0) {
		if (commandTokens.size() >= 5) {
			try {
				int indx = stoi(commandTokens[2]);
				int indy = stoi(commandTokens[3]);
				string prop = commandTokens[4];
				double newVal = stod(commandTokens[5]);

				bool err = false;
				if (indx < 0 || indx >= trapControllerHandler.statHandler.x->traps.size()) {
					cout << "Index out of range!" << endl;
					err = true;
				}
				else if (prop.compare("freq") != 0 && prop.compare("amp") != 0 && prop.compare("phase") != 0) {
					cout << "Must select property freq, amp, or phase!" << endl;
					err = true;
				}

				if (err) {
					cout << "Usage: change [trap index] [property: freq, amp, or phase] [new value]" << endl;
				}
				else {
					waveformShouldChange = true;
					if (prop.compare("freq") == 0) {
						trapControllerHandler.statHandler.x->traps[indx].frequency = newVal * 1.0E6;
					}
					else if (prop.compare("amp") == 0) {
						trapControllerHandler.statHandler.x->traps[indx].amplitude = newVal;
					}
					else if (prop.compare("phase") == 0) {
						trapControllerHandler.statHandler.x->traps[indx].setPhase(newVal);
					}
				}
			}
			catch (const std::invalid_argument&) {
				cout << "Unable to parse values!" << endl;
			}
		}
		else {
			cout << "Usage: change [trap index] [property: freq,amp,phase] [new value]" << endl;
		}
	}
	else if (commandTokens[1].compare("load_default") == 0) {
		if (commandTokens.size() >= 3) {
			string configuration_filename = commandTokens[2];
			if (trapControllerHandler.loadDefaultTrapConfiguration(configuration_filename)) {
				//continue;

				awgController.allocateDynamicWFBuffer(m_dur, trapControllerHandler.statHandler.x->traps.size(),trapControllerHandler.statHandler.x->traps.size());
				cout << "Loading precomputed waveforms from disk." << endl;

				// Default name of trap configurations to start and end with.
				string starting_configuration = trapControllerHandler.lastLoadedConfiguration;
				string ending_configuration = trapControllerHandler.lastLoadedConfiguration;

				if (!trapControllerHandler.loadPrecomputedWaveforms(m_dur, starting_configuration, ending_configuration)){
					return false;
				}
			}
		}
		else {
			cout << "Usage: load_default [default_name]" << endl;
			trapControllerHandler.printAvailableDefaultTrapConfigurations();
		}
	}
	else if (commandTokens[1].compare("set_phases") == 0) {
		if (commandTokens.size() < 3) {
			cout << "Usages:" << endl;
			cout << "Random phases: set_phases random" << endl;
			cout << "Cyclic pattern: set_phases pattern [phase1] [phase2] [phase3] ..." << endl;
			cout << "Phase step: set_phases step [first_phase] [phase_step]" << endl;
		}
		else if (commandTokens[2].compare("random") == 0) {
			waveformShouldChange = true;
			for (int i = 0; i < trapControllerHandler.statHandler.x->traps.size(); i++) {
				double phase = (rand() % 10000) / 10000.0;
				trapControllerHandler.statHandler.x->traps[i].setPhase(phase);
			}
		}
	else if (commandTokens[2].compare("pattern") == 0) {
			// try {
			// 	// Load pattern vector.
			// 	vector<double> phasePattern;
			// 	for (int i = 3; i < commandTokens.size(); i++) {
			// 		phasePattern.push_back(stod(commandTokens[i]));
			// 	}
			//
			// 	// If no arguments provided, set all to 0.
			// 	if (phasePattern.size() == 0) {
			// 		phasePattern.push_back(0);
			// 	}
			//
			// 	// Update phases.
			// 	for (int i = 0; i < trapController.traps.size(); i++) {
			// 		int phasePatternIndex = i % phasePattern.size();
			// 		trapController.traps[i].setPhase(phasePattern[phasePatternIndex]);
			// 	}
			// 	waveformShouldChange = true;
			//
			// }
			// catch (const std::invalid_argument&) {
			// 	cout << "Unable to parse." << endl;
			// }
		}
	else if (commandTokens[2].compare("step") == 0) {
		// try {
		// 	if (commandTokens.size() < 5) {
		// 		cout << "Usage: set_phases step [offset] [step]" << endl;
		// 	}
		// 	else {
		// 		double offset = stod(commandTokens[3]);
		// 		double step = stod(commandTokens[4]);
		//
		// 		double phase = offset;
		// 		for (int i = 0; i < trapController.traps.size(); i++) {
		// 			trapController.traps[i].setPhase(phase);
		// 			phase += step * i;
		// 			while (phase >= 1.0) {
		// 				phase -= 1.0;
		// 			}
		// 		}
		// 		waveformShouldChange = true;
		//
		// 	}
		// }
		// catch (const std::invalid_argument&) {
		// 	cout << "Unable to parse." << endl;
		// }
	}
	else {
			cout << "Unable to parse: " << commandTokens[1] << endl;
		}
	}
	else {
		if (commandTokens[1].compare("help") != 0) {
			cout << commandTokens[1] << ": trap command not found!" << endl;
		}
		printTrapHelp();
	}
	if (waveformShouldChange) {
		trapControllerHandler.sanitizeTraps();
		waveformShouldChange = false;
		}
		return waveformShouldChange;
}

bool process2DInput(std::vector<string> &commandTokens, TrapControllerHandler &trapControllerHandler, AWGController &awgController) {
	string mainCommand = commandTokens[0];

	if (mainCommand.compare("traps") == 0) {
		bool waveformShouldBeRecalculated = processTrapsInput(commandTokens, trapControllerHandler, awgController);
		if (waveformShouldBeRecalculated && awgController.isConnected()) {
				awgController.pushStaticWaveforms(trapControllerHandler.generateStaticWaveform(),true);
		}
 } else if (mainCommand.compare("awg") == 0) {
	 	processAWGInput(commandTokens, trapControllerHandler, awgController);
	} else if (mainCommand.compare("run") == 0) {
		processRunCommand(commandTokens, trapControllerHandler, awgController);
	// } else if (mainCommand.compare("adwin") == 0) {
	// 	processAdwinCommand(commandTokens);
	} else if (mainCommand.compare("help") == 0) {
		cout << "Console help menu:" << endl;
		cout << "traps [trap command] [arguments]" << endl;
		cout << "awg [awg command] [arguments]" << endl;
		cout << "run [run command] [arguments]" << endl;
		cout << "adwin [adwin command]" << endl;
	} else if (mainCommand.compare("exit") == 0) {
		awgController.disconnect();
		for(int i = 0;i<trapControllerHandler.tchLen;i++){
			for(int j = 0;j<trapControllerHandler.tchLen;j++){
				//free(trapControllerHandler.statHandler.x->loadedTrapWaveforms[i][j]);
				hipFree(trapControllerHandler.statHandler.x->loadedCudaWaveforms[i][j]);
				if(numDevices == 2){
					hipFree(trapControllerHandler.statHandler.x->loadedCudaWaveforms2[i][j]);
				}
			}
		}
		awgController.cleanCudaBuffer();
		trapControllerHandler.cleanCudaModes();
		cout << "Bye!" << endl;
		return true;
	} else {
		cout << "Command unknown: " << commandTokens[0] << endl;
		cout << "Type 'help' for more information." << endl;
	}

	return false;
}

void run2DConsole( TrapControllerHandler trapControllerHandler, AWGController &awgController) {

	bool shouldExit = false;
	while (!shouldExit) {
			std::vector<string> commandTokens = takeInput();
		if (commandTokens.size() == 0) {
			continue;
		}
		shouldExit = process2DInput(commandTokens, trapControllerHandler, awgController);
	}
}
