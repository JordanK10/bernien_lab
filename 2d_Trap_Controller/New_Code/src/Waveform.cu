// Created by H.Levine with Atom Array team, Lukin group (2016).
// Adapted by J.Kemp with Atom Array team, Bernien Group (2019).



#include "Waveform.h"
#include <iostream>
#include <fstream>
#include <iterator>
#include <iomanip>

using namespace std;

Waveform::Waveform() {
}

Waveform::Waveform(string binaryFilename) {
	initializeFromBinaryFile(binaryFilename);
}

Waveform::Waveform(std::vector<short>waveform){
	dataVectorShort = waveform;
}

bool Waveform::fileExists(string filename) {
	string path(filename);

	ifstream file(path, ios::in | ios::binary);
	return file.good();
}

Waveform::Waveform(short* data)
{
	dataShort = data;
}



int Waveform::initializeFromBinaryFile(string binaryFilename) {
	string path(dir);
	path.append(binaryFilename);
		ifstream binaryFile(path, ios::in | ios::binary);
		unsigned int numElements;
		binaryFile.read(reinterpret_cast<char*>(&numElements), sizeof(unsigned int));
		if(numElements%32 != 0){numElements --;}
		dataShort = static_cast<short*>(malloc(sizeof(short)*numElements));
		binaryFile.read(reinterpret_cast<char*>(&dataShort[0]), numElements * sizeof(short));
		return numElements;
}


bool Waveform::initializeFromStaticWaveform(string trap_configuration_file) {
	int length = trap_configuration_file.length();
	string waveform_filename = trap_configuration_file.substr(0, length - 4) + "_static"; // Remove .txt, replace with _static.
	string waveform_path = static_waveforms + waveform_filename;
	if (fileExists(waveform_path)) {
		initializeFromBinaryFile(waveform_path);
		return true;
	} else {
		return false;
	}
}

int Waveform::initializeFromMovingWaveform(double duration,
											string starting_configuration, string ending_configuration,
											int start_index, int end_index) {
	int starting_length = starting_configuration.length();
	int ending_length = ending_configuration.length();

	stringstream dirStream;
	dirStream << starting_configuration.substr(0, starting_length - 4) << "_to_";
	dirStream << ending_configuration.substr(0, ending_length - 4) << "_in_";
	dirStream << fixed << setprecision(1) << duration << "ms/";

	stringstream fileStream;
	fileStream << "rearrange_" << start_index << "_to_" << end_index;


	string path = dirStream.str() + fileStream.str();
	return initializeFromBinaryFile(path);
}

void Waveform::writeToFile(string filename) {
	// string path(dir);
	// path.append(filename);
	//
	// ofstream file(path);
	// std::ostream_iterator<complex<float>> output_iterator(file, "\n");
	//
	// copy(std::begin(dataVector), std::end(dataVector), output_iterator);
	return;
}


void Waveform::writeToBinaryFile(string filename) {
	// string path(dir);
	// path.append(filename);
	//
	// ofstream file(path, ios::out | ios::binary);
	// const char *dataPointer = reinterpret_cast<const char*>(&dataVector[0]);
	// unsigned int numElements = dataVector.size();
	//
	// file.write(reinterpret_cast<const char*>(&numElements), sizeof(unsigned int));
	// file.write(dataPointer, numElements * sizeof(dataVector[0]));
	// file.close();
	return;
}
